
#include <hip/hip_runtime.h>
extern "C"{
__global__ void applyFilter(const unsigned char *inputChannel, unsigned char *outputChannel,
                             const unsigned int width, const unsigned int height,
                             const unsigned int filterWidth) {
    unsigned int y;
    unsigned int x;
    unsigned int blur;
    int filterHalf;
    unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    if(row < height && col < width) {
        // Unwrapped/optimized versions for small filters
        if(filterWidth == 1){ // 1x1 filter (Do nothing)
            outputChannel[col + row * width] = inputChannel[col + row * width];
            return;
        }else if(filterWidth == 3 && row > 0 && row < height -1 && col > 0 && col < width - 1){ // 3x3 filter
            blur = 0;
            blur += inputChannel[col + 1 + row * width + width];
            blur += inputChannel[col + 1 + row * width];
            blur += inputChannel[col + 1 + row * width - width];
            blur += inputChannel[col + row * width + width];
            blur += inputChannel[col + row * width];
            blur += inputChannel[col + row * width - width];
            blur += inputChannel[col - 1 + row * width + width];
            blur += inputChannel[col - 1 + row * width];
            blur += inputChannel[col - 1 + row * width - width];
            outputChannel[col + row * width] = blur / 9;
        }


        filterHalf = filterWidth / 2;
        blur = 0;
        for(int i = -filterHalf; i <= filterHalf; i++) {
            for(int j = -filterHalf; j <= filterHalf; j++) {
                y = max(0, min(height - 1, row + i));
                x = max(0, min(width - 1, col + j));
                blur += inputChannel[x + y * width];
            }
        }
        outputChannel[col + row * width] = blur / (filterWidth * filterWidth);
    }
}
}